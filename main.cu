#include "hip/hip_runtime.h"
#include "rt_weekend.h"

#include "camera.h"
#include "disk.h"
#include "hittable.h"
#include "hittable_list.h"
#include "interval.h"
#include "material.h"
#include "sphere.h"

#include <hiprand/hiprand_kernel.h>

#include <chrono>
#include <fstream>
#include <iostream>

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file,
                int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":"
                  << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void create_world(hittable **world, int n_objects, hiprandState *rs) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rs);

        // material *mat_ground = new lambertian(color(0.8, 0.8, 0.8));
        material *mat_ground = new metal(color(0.5, 0.5, 0.5), 0.3);
        material *mat1 = new lambertian(color(0.1, 0.2, 0.5));
        material *mat2 = new dielectric(1.5);
        material *mat3 = new metal(color(0.8, 0.6, 0.2), 0);
        
        hittable **objects = new hittable*[n_objects];

        // These dereferenced assignments to dynamic objects require double pointers.
        objects[0] = new disk(ray(point3(0, 0, 0), vec3(0, 1, 0)), 10, mat_ground);
        objects[1] = new sphere(point3(-3.5, 1, -0.8), 1, mat1);
        objects[2] = new sphere(point3(-0.5, 1, 2), 1, mat2);
        objects[3] = new sphere(point3(2.5, 1, 1.5), 1, mat3);
        
        for (int i = 4; i < n_objects; i++) {
            float choose_mat = hiprand_uniform(rs);
            point3 center(hiprand_uniform(rs)*10-5, 0.2, hiprand_uniform(rs)*10-5);

            if (choose_mat < 1.0/3.0) {
                color albedo = color::random(rs);
                material *mat = new lambertian(albedo);
                objects[i] = new sphere(center, 0.2, mat);
            } else if (choose_mat < 2.0/3.0) {
                color albedo = color::random(0.5, 1, rs);
                float fuzz = hiprand_uniform(rs) / 2;
                material *mat = new metal(albedo, fuzz);
                objects[i] = new sphere(center, 0.2, mat);
            } else {
                material *mat = new dielectric(1.5);
                objects[i] = new sphere(center, 0.2, mat);
            }
        }

        *world = new hittable_list(objects, n_objects);
    }
}

__global__ void free_world(hittable **world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        delete *world;
    }
}

__global__ void init_camera(camera **cam, int iw, int ih, float vfov, point3 look_from,
                            point3 look_at, vec3 vup, float defocus_angle, float focus_dist) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *cam = new camera(iw, ih, vfov, look_from, look_at, vup, defocus_angle, focus_dist);
    }
}

__global__ void free_camera(camera **cam) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        delete *cam;
    }
}

__global__ void init_render(hiprandState *rand_state, int iw, int ih) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= iw) || (j >= ih))
        return;

    int pixel_index = j * iw + i;
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__device__ color ray_color(const ray &r, const hittable **world, hiprandState *rs) {
    ray cur_ray = r;
    color cur_attenuation = color(1.0, 1.0, 1.0);
    bool debug = false;
    for (int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, interval(0.001, INFINITY), rec)) {
            ray scattered;
            color attenuation;
            if (rec.mat->scatter(cur_ray, rec, attenuation, scattered, rs, debug)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            } else {
                return color(0, 0, 0);
            }
        } else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float a = 0.5 * (unit_direction.y() + 1.0);
            color bg = (1.0 - a) * color(1.0, 1.0, 1.0) + a * color(0.5, 0.7, 1.0);
            return cur_attenuation * bg;
        }
    }
}

__global__ void render(color *fb, int iw, int ih, int n_samples, camera **cam, hittable **world,
                       hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= iw) || (j >= ih))
        return;

    int pixel_index = j * iw + i;
    hiprandState rs = rand_state[pixel_index];
    color c(0.0f, 0.0f, 0.0f);
    for (int s = 0; s < n_samples; s++) {
        ray r = (*cam)->get_ray(i, j, &rs);
        c += ray_color(r, const_cast<const hittable **>(world), &rs);
    }
    fb[pixel_index] = c / float(n_samples);
}

int main() {
    // Image

    float aspect_ratio = 16.0 / 9.0;
    int image_width = 1200;
    int samples_per_pixel = 500;

    // Calculate the image height, and ensure that it's at least 1.
    int image_height = int(image_width / aspect_ratio);
    image_height = (image_height < 1) ? 1 : image_height;
    int num_pixels = image_width * image_height;

    // Create world

    int n_objects = 50;

    hittable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));

    hiprandState *d_world_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_world_rand_state, sizeof(hiprandState)));

    create_world<<<1, 1>>>(d_world, n_objects, d_world_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Create camera

    // Camera configuration parameters.
    float vfov = 40;
    point3 look_from(0, 1.5, 8);
    point3 look_at(0, 1, 0);
    vec3 vup(0, 1, 0);
    float defocus_angle = 1.0;
    float focus_dist = 6;

    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));

    init_camera<<<1, 1>>>(d_camera, image_width, image_height, vfov, look_from, look_at, vup,
                          defocus_angle, focus_dist);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Render

    auto start = std::chrono::high_resolution_clock::now();

    // Allocate frame buffer
    size_t fb_size = 3 * num_pixels * sizeof(float);
    color *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    int tx = 16;
    int ty = 16;

    // Allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));

    // Render
    dim3 blocks((image_width + tx - 1) / tx, (image_height + ty - 1) / ty);
    dim3 threads(tx, ty);
    std::cout << "blocks.x = " << blocks.x << std::endl;
    std::cout << "blocks.y = " << blocks.y << std::endl;
    std::cout << "threads.x = " << threads.x << std::endl;
    std::cout << "threads.y = " << threads.y << std::endl;

    init_render<<<blocks, threads>>>(d_rand_state, image_width, image_height);
    render<<<blocks, threads>>>(fb, image_width, image_height, samples_per_pixel, d_camera, d_world,
                                d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start) / 1e6;

    // Write output image

    std::ofstream file;
    std::string file_name = "image.ppm";
    file.open(file_name, std::ios::trunc);

    if (file.is_open()) {
        // Output FB as .ppm image
        file << "P3\n" << image_width << " " << image_height << "\n255\n";
        for (int j = 0; j < image_height; j++) {
            for (int i = 0; i < image_width; i++) {
                size_t pixel_index = j * image_width + i;
                color pixel_color = fb[pixel_index];
                write_color(file, pixel_color);
            }
        }
        file.close();
        std::cout << "Successfully wrote to " << file_name << std::endl;
    } else {
        std::cerr << "Unable to open file: " << file_name << std::endl;
    }

    // Free memory
    free_world<<<1, 1>>>(d_world);
    checkCudaErrors(hipGetLastError());
    free_camera<<<1, 1>>>(d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_world_rand_state));
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));

    std::cout << "Rendered in " << duration.count() << " seconds" << std::endl;
}
