#include "hip/hip_runtime.h"
#include "constants.h"
#include "camera.h"
#include "hittable.h"
#include "hittable_list.h"
#include "interval.h"
#include "sphere.h"

#include <hiprand/hiprand_kernel.h>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render_init(hiprandState *rand_state, int iw, int ih) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= iw) || (j >= ih)) return;
    
    int pixel_index = j*iw + i;
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void create_world(hittable **obj_list, hittable **world, camera **cam, int iw, int ih) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        // These dereferenced assignments to dynamic objects require double pointers.
        *(obj_list) = new sphere(point3(0, 0, -1), 0.5);
        *(obj_list+1) = new sphere(point3(0, -100.5, -1), 100);
        *world = new hittable_list(obj_list, 2);
        *cam = new camera(iw, ih);
    }
}

__device__ color ray_color(const ray& r, const hittable **world) {
    hit_record rec;
    if ((*world)->hit(r, interval(0, INFINITY), rec)) {
        return 0.5f * (rec.normal + color(1.0f, 1.0f, 1.0f));
    }
    vec3 unit_direction = unit_vector(r.direction());
    float a = 0.5f * (unit_direction.y() + 1.0f);
    return (1.0f-a)*color(1.0f, 1.0f, 1.0f) + a*color(0.5f, 0.7f, 1.0f);
}

__global__ void render(
    color *fb,
    int iw,
    int ih,
    int n_samples,
    camera **cam,
    hittable **world,
    hiprandState *rand_state
) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= iw) || (j >= ih)) return;

    int pixel_index = j*iw + i;
    hiprandState rs = rand_state[pixel_index];
    color c(0.0f, 0.0f, 0.0f);
    for (int s = 0; s < n_samples; s++) {
        // float u = float(i + hiprand_uniform(&rands)) / float(iw);
        // float v = float(j + hiprand_uniform(&rands)) / float(ih);
        ray r = (*cam)->get_ray(i, j, &rs);
        c += ray_color(r, const_cast<const hittable **>(world));
    }
    fb[pixel_index] = c / float(n_samples);
}

int main() {
    // Image

    float aspect_ratio = 16.0f / 9.0f;
    int image_width = 400;
    int samples_per_pixel = 100;

    // Calculate the image height, and ensure that it's at least 1.
    int image_height = int(image_width / aspect_ratio);
    image_height = (image_height < 1) ? 1 : image_height;
    int num_pixels = image_width * image_height;

    // Create world

    hittable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 2*sizeof(hittable *)));

    hittable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));

    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));

    create_world<<<1, 1>>>(d_list, d_world, d_camera, image_width, image_height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Render

    // Allocate frame buffer
    size_t fb_size = 3 * num_pixels * sizeof(float);
    color *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));
    
    int tx = 16;
    int ty = 16;

    // Allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));

    // Render our buffer
    dim3 blocks((image_width+tx-1)/tx, (image_height+ty-1)/ty);
    dim3 threads(tx, ty);
    std::clog << "blocks.x = " << blocks.x << "\n";
    std::clog << "blocks.y = " << blocks.y << "\n";
    std::clog << "threads.x = " << threads.x << "\n";
    std::clog << "threads.y = " << threads.y << "\n";

    render_init<<<blocks, threads>>>(d_rand_state, image_width, image_height);
    render<<<blocks, threads>>>(fb, image_width, image_height, samples_per_pixel, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    
    // Output FB as .ppm image
    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = 0; j < image_height; j++) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j*image_width + i;
            color pixel_color = fb[pixel_index];
            write_color(std::cout, pixel_color);
        }
    }

    checkCudaErrors(hipFree(fb));
    std::clog << "Done.\n";
}
